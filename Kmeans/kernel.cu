#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "Kmeans.h"

#define CHKMAL_ERROR	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
#define CHKMEMCPY_ERROR if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
#define CHKSYNC_ERROR	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize failed! Error code %d\n", cudaStatus); goto Error; }

// arrSize indices; THREADS_PER_BLOCK * NO_BLOCKS total threads;
// Each thread in charge of THREAD_BLOCK_SIZE contigeous indices
     
#define THREADS_PER_BLOCK 1000

__global__ void reClusterWithCuda(xyArrays* d_kCenters, xyArrays* d_xya, const int size, bool *kaFlags)
{
	extern __shared__ bool* d_kaFlags; // array to flag changes in point-to-cluster association

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	d_kaFlags[i] = false;
	
	// for every point: save idx where min(distance from k[idx])
	//#pragma omp parallel for reduction(|:kAssociationChangedFlag)
	
	if (i < size) {
		int prevPka = d_kCenters[i]; // save associated cluster idx
		for (int idx = 0; idx < size; idx++)
		{

		} 
		
		tmpx = data->x[i];
		float tmpy = data->y[i];
		tmpx += 10.f;
		tmpy += 20.f;
		result->x[i] = tmpx;
		result->y[i] = tmpy;
	}



		int prevPka = pka[i];  // save associated cluster idx
		getNewPointKCenterAssociation(i, size);
		if (pka[i] != prevPka)
		{
			kaFlag = true;
		}



	//c[i] = a[i] + b[i];
}

// Helper function for finding best centers for ksize clusters
hipError_t kCentersWithCuda(xyArrays* kCenters, xyArrays* xya, long N, int ksize, int LIMIT)
{
	hipError_t cudaStatus; 
	const int NO_BLOCKS = N / THREADS_PER_BLOCK;
	const int THREAD_BLOCK_SIZE = N / (THREADS_PER_BLOCK * NO_BLOCKS);
	if (N % (THREADS_PER_BLOCK * NO_BLOCKS) != 0) {
		fprintf(stderr, "reClusterWithCuda launch failed:\n"
			"Array size (%d) modulo Total threads (%d) != 0.\n"
			"Try changing number of threads.\n", N, (THREADS_PER_BLOCK * NO_BLOCKS));
		goto Error;
	}

	// memory init block
	{
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		size_t nDataBytes = sizeof(xya);
		size_t nKCenterBytes = sizeof(kCenters);

		// allocate device memory
		xyArrays *d_a, *d_k;		// data and k-centers xy information
		int* d_pka;
		
									//bool *d_kaFlags;			// array to flag changes in point-to-cluster association

		hipMalloc((xyArrays**)&d_a, nDataBytes); CHKMAL_ERROR;
		hipMalloc((xyArrays**)&d_k, nKCenterBytes); CHKMAL_ERROR;
		hipMalloc((int**)&d_pka, N * sizeof(int)); CHKMAL_ERROR;
		//hipMalloc((bool**)&d_kaFlags, N * sizeof(bool)); CHKMAL_ERROR;

		initK(ksize);				// K-centers = first points in data (on host)

									// copy data from host to device
		hipMemcpy(d_a, xya, nDataBytes, hipMemcpyHostToDevice); CHKMEMCPY_ERROR;
		hipMemcpy(d_k, kCenters, nKCenterBytes, hipMemcpyHostToDevice); CHKMEMCPY_ERROR;

		//cudaStatus = hipMemset((void*)dev_threadedHist, 0, THREADS_PER_BLOCK * NO_BLOCKS * histSize * sizeof(int));
		cudaStatus = hipMemset((void*)d_kaFlags, 0, N * sizeof(bool));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemset failed!\n");
			goto Error;
		}
	}
	

	// *** phase 1 ***
	// One thread for every THREAD_BLOCK_SIZE elements.

	reClusterWithCuda << <NO_BLOCKS, THREADS_PER_BLOCK >> >(d_k, d_a, d_kaFlags, THREADS_PER_BLOCK, THREAD_BLOCK_SIZE);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "threadedHistKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize(); CHKSYNC_ERROR;
	

	
	

	
	int iter = 0;
	bool kAssociationChangedFlag = true;
	do {
		//printf("iter %d\n", iter + 1);
		reClusterWithCuda(ksize, kAssociationChangedFlag);
	} while (++iter < LIMIT && kAssociationChangedFlag);  // association changes: need to re-cluster

	//TODO quick test
	for (int i = 0; i < ksize; i++)
	{
		printf("%d, %f, %f\n", i, kCenters->x[i], kCenters->y[i]);
	}

	//float x = input[threadID];
	//float y = func(x);
	//output[threadID] = y;
	Error:
		hipFree(d_a);
		hipFree(d_k);

		return cudaStatus;
}




/**************/
//old cuda code
/*
	__global__ void addKernel(int *c, const int *a, const int *b)
	{
		int i = threadIdx.x;
		c[i] = a[i] + b[i];
	}

	// Helper function for using CUDA to add vectors in parallel.
	hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
	{
		int *dev_a = 0;
		int *dev_b = 0;
		int *dev_c = 0;
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// Launch a kernel on the GPU with one thread for each element.
		addKernel << <1, size >> >(dev_c, dev_a, dev_b);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

	Error:
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		return cudaStatus;
	}

	*/